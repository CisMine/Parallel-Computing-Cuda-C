
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void staticReverse(int *d, int n)
{
  __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

__global__ void dynamicReverse(int *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64;
  int a[n], r[n], d[n];

  for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }

  int *d_d;
  hipMalloc(&d_d, n * sizeof(int)); 

  // run version with static shared memory
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  staticReverse<<<1,n>>>(d_d, n);
  hipMemcpy(d, d_d, n*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);

  // run dynamic shared memory version
  hipMemcpy(d_d, a, n*sizeof(int), hipMemcpyHostToDevice);
  dynamicReverse<<<1,n,n*sizeof(int)>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) 
    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)n", i, i, d[i], r[i]);
}
